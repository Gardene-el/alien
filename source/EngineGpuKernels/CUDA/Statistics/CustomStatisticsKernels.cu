#include "hip/hip_runtime.h"
#include "CustomStatisticsKernels.cuh"


__global__ void cudaUpdateHeatmap_substep1(SimulationData data, SimulationStatistics statistics){
    statistics.addExternalEnergy(*data.externalEnergy);
}
__global__ void cudaUpdateHeatmap_substep2(SimulationData data, SimulationMapStatistics statistics){

    statistics.resetMaps();
    {
        auto& cells = data.objects.cellPointers;
        auto const partition = calcAllThreadsPartition(cells.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);
            statistics.incNumCells(cell->pos);
        }
    }
}
__global__ void cudaUpdateHeatmap_substep3(SimulationData data, SimulationMapStatistics statistics){
}