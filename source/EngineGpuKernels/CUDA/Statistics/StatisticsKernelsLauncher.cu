﻿#include "StatisticsKernelsLauncher.cuh"

#include "StatisticsKernels.cuh"

#include "CustomStatisticsKernels.cuh"

void _StatisticsKernelsLauncher::updateStatistics(GpuSettings const& gpuSettings, SimulationData const& data, SimulationStatistics const& simulationStatistics)
{
    KERNEL_CALL_1_1(cudaUpdateTimestepStatistics_substep1, data, simulationStatistics);
    KERNEL_CALL(cudaUpdateTimestepStatistics_substep2, data, simulationStatistics);
    KERNEL_CALL_1_1(cudaUpdateTimestepStatistics_substep3, data, simulationStatistics);
    
    KERNEL_CALL_1_1(cudaUpdateHistogramData_substep1, data, simulationStatistics);
    KERNEL_CALL(cudaUpdateHistogramData_substep2, data, simulationStatistics);
    KERNEL_CALL(cudaUpdateHistogramData_substep3, data, simulationStatistics);
}
    void _StatisticsKernelsLauncher::updateCustomStatistics(GpuSettings const& gpuSettings, SimulationData const& data, SimulationStatistics const& simulationStatistics,SimulationMapStatistics const&simulationMapStatistics){
        
    KERNEL_CALL_1_1(cudaUpdateHeatmap_substep1, data, simulationStatistics);

    KERNEL_CALL(cudaUpdateHeatmap_substep2, data, simulationMapStatistics);
    }
