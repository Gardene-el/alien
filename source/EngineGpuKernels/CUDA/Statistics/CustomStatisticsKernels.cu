#include "hip/hip_runtime.h"
#include "CustomStatisticsKernels.cuh"


__global__ void cudaUpdateHeatmap_substep1(SimulationData data, SimulationStatistics statistics){
    statistics.addExternalEnergy(*data.externalEnergy);
}
__global__ void cudaUpdateHeatmap_substep2(SimulationData data, SimulationMapStatistics statistics){
}
